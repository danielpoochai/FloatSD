
#include <hip/hip_runtime.h>
#define FLOAT_TO_BITS(x) (*reinterpret_cast<unsigned int*>(x))
#define BITS_TO_FLOAT(x) (*reinterpret_cast<float*>(x))

__device__ __forceinline__ unsigned int extract_exponent(float *a) {
  unsigned int temp = *(reinterpret_cast<unsigned int*>(a));
  temp = (temp << 1 >> 24); // single preciision, 1 sign bit, 23 mantissa bits
  return temp-127+1; // exponent offset and virtual bit
}

__device__ __forceinline__ unsigned int round_bitwise_stochastic(unsigned int target,
                                                                 unsigned int rand_prob){
    int man_bits = 5;
    unsigned int mask = (1 << (23-man_bits)) - 1;
    unsigned int add_r = target+(rand_prob & mask);
    unsigned int quantized = add_r & ~mask;
    return quantized;
}

__device__ __forceinline__ unsigned int round_bitwise_nearest(unsigned int target) {
    int man_bits = 5;
    unsigned int mask = (1 << (23-man_bits)) - 1;
    unsigned int rand_prob = 1 << (23-man_bits-1);
    unsigned int add_r = target+rand_prob;
    unsigned int quantized = add_r & ~mask;
    return quantized;
}

__device__ __forceinline__ unsigned int clip_exponent(unsigned int old_num,
                                                      unsigned int quantized_num) {
  int exp_bits = 5;
  int man_bits = 2;
  int quantized_exponent_store = quantized_num << 1 >> 1 >> 23; // 1 sign bit, 23 mantissa bits
  // int min_exponent_store = -((1 << (exp_bits-1))-1) + 127;
  // int max_exponent_store = ((1 << (exp_bits-1))-1) + 127; 
  int min_exponent_store = -(24) + 127;
  int max_exponent_store = (7) + 127; 
  if (quantized_exponent_store > max_exponent_store) {
    unsigned int max_man = (unsigned int ) -1 << 9 >> 9 >> (23-man_bits) << (23-man_bits); // 1 sign bit, 8 exponent bits, 1 virtual bit
    unsigned int max_num = ((unsigned int) max_exponent_store << 23) | max_man;
    unsigned int old_sign = old_num >> 31 << 31;
    quantized_num = old_sign | max_num;
  } else if (quantized_exponent_store < min_exponent_store) {
    unsigned int min_num = ((unsigned int) min_exponent_store << 23);
    unsigned int old_sign = old_num >> 31 << 31;
    quantized_num = old_sign | min_num;
  }
  return quantized_num;
}


__device__ __forceinline__ unsigned int clip_max_exponent(int man_bits,
                                                          unsigned int max_exponent,
                                                          unsigned int quantized_num) {
  unsigned int quantized_exponent = quantized_num << 1 >> 24 << 23; // 1 sign bit, 23 mantissa bits
  if (quantized_exponent > max_exponent) {
    unsigned int max_man = (unsigned int ) -1 << 9 >> 9 >> (23-man_bits) << (23-man_bits); // 1 sign bit, 8 exponent bits
    unsigned int max_num = max_exponent | max_man;
    unsigned int old_sign = quantized_num >> 31 << 31;
    quantized_num = old_sign | max_num;
  }
  return quantized_num;
}
